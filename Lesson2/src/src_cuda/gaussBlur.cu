#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include "gaussBlur.cuh"
using  namespace std;
using namespace cv;


__global__  void  separate_channels(const uchar4* const d_inputImageRGBA,unsigned char* const d_red,
                                                                        unsigned char* const d_green,
                                    					                unsigned char* const d_blue,
                                    					                int rows,int cols)
{
    const int r=blockIdx.y*blockDim.y+threadIdx.y;
    const int c=blockIdx.x*blockDim.x+threadIdx.x;
    int idx=r*cols+c;

    if(r>=rows||c>=cols)
    {
        return;
    }

    d_red[idx]=d_inputImageRGBA[idx].x;
    d_green[idx]=d_inputImageRGBA[idx].y;
    d_blue[idx]=d_inputImageRGBA[idx].z;

}

__global__ void gaussblur(const unsigned char* const d_red, unsigned char* const d_redBlurred ,
                           float* d_filter,int rows,int cols,const int filterWidth)
{
    const int r=blockIdx.y*blockDim.y+threadIdx.y;
    const int c=blockIdx.x*blockDim.x+threadIdx.x;
    const int idx=r*cols+c;

    if(r>=rows||c>=cols)
    {
            return;
    }

	float color = 0.0f;
	for (int filter_y = 0; filter_y < filterWidth; filter_y++) {
		for (int filter_x = 0; filter_x < filterWidth; filter_x++) {
			int image_x = c + filter_x - filterWidth / 2;
			int image_y = r + filter_y - filterWidth / 2;
			image_x  = min(max(image_x , 0), cols - 1);
			image_y  = min(max(image_y , 0), rows - 1);

			float filter_value = d_filter[filter_y*filterWidth + filter_x];
			color += filter_value * static_cast<float>(d_red[image_y *cols + image_x]);
		}
	}
	d_redBlurred[idx] = color;

}

__global__ void recombineChannels(const unsigned char* const d_redBlurred,const unsigned char* const d_greenBlurred,
                                  const unsigned char* const d_blueBlurred, uchar4* const d_outputImageRGBA,
                                  int rows,int cols)
{
     const int r=blockIdx.y*blockDim.y+threadIdx.y;
     const int c=blockIdx.x*blockDim.x+threadIdx.x;
     const int idx=r*cols+c;

     if(r>=rows||c>=cols)
     {
           return;
     }

    unsigned char red = d_redBlurred[idx];
	unsigned char green = d_greenBlurred[idx];
	unsigned char blue = d_blueBlurred[idx];

	uchar4 outputPixel = make_uchar4(red, green, blue, 255);

	d_outputImageRGBA[idx] = outputPixel;
}


cv::Mat gauss_blur(cv::Mat inputImage,cv::Mat inputTMP,const float *h_filter, int filterWidth)
{
    uchar4 *h_inputImageRGBA,*h_outputImage;
    uchar4 *d_inputImageRGBA,*d_outputImageRGBA;
    unsigned char  *d_red,*d_green,*d_blue,*d_redBlurred,*d_greenBlurred,*d_blueBlurred;

    float *d_filter;

    h_inputImageRGBA=(uchar4 *)inputImage.ptr<unsigned char>(0);
    h_outputImage=(uchar4 *)inputTMP.ptr<unsigned char>(0);


    const int rows=inputImage.rows;
    const int cols=inputImage.cols;
    const size_t numPixels=rows*cols;

    //分配显存
    hipMalloc((void **)&d_inputImageRGBA,sizeof(uchar4)*numPixels);
    hipMalloc((void **)&d_outputImageRGBA,sizeof(uchar4)*numPixels);
    hipMalloc((void **)&d_red,sizeof(unsigned char)*numPixels);
    hipMalloc((void **)&d_green,sizeof(unsigned char)*numPixels);
    hipMalloc((void **)&d_blue,sizeof(unsigned char)*numPixels);
    hipMalloc((void **)&d_redBlurred,sizeof(unsigned char)*numPixels);
    hipMalloc((void **)&d_greenBlurred,sizeof(unsigned char)*numPixels);
    hipMalloc((void **)&d_blueBlurred,sizeof(unsigned char)*numPixels);
    hipMalloc((void **)&d_filter,sizeof(float)*filterWidth*filterWidth);

    //将CPU中h_inputImageRGBA的数据复制到GPU中d_inputImageRGBA
    hipMemcpy(d_inputImageRGBA,h_inputImageRGBA,sizeof(uchar4)*numPixels,hipMemcpyHostToDevice);
    //将CPU中h_filter的数据复制到GPU中d_filter
    hipMemcpy(d_filter,h_filter,sizeof(float)*filterWidth*filterWidth,hipMemcpyHostToDevice);

    const dim3 blockSize(32,32,1);
    const dim3 gridSize((cols-1+32)/32,(rows-1+32)/32,1);

    //将输入图片分离为三个颜色通道
    separate_channels<<<gridSize,blockSize>>>(d_inputImageRGBA,d_red,d_green,d_blue,rows,cols);
    hipDeviceSynchronize();

    //对单颜色通道进行高斯模糊
    gaussblur<<<gridSize,blockSize>>>(d_red,d_redBlurred,d_filter,rows,cols,filterWidth);
    hipDeviceSynchronize();
    gaussblur<<<gridSize,blockSize>>>(d_green,d_greenBlurred,d_filter,rows,cols,filterWidth);
    hipDeviceSynchronize();
    gaussblur<<<gridSize,blockSize>>>(d_blue,d_blueBlurred,d_filter,rows,cols,filterWidth);
    hipDeviceSynchronize();

    //合并高斯模糊后的三条颜色通道
    recombineChannels<<<gridSize,blockSize>>>(d_redBlurred,d_greenBlurred,d_blueBlurred,d_outputImageRGBA,rows,cols);
    hipDeviceSynchronize();

    //将GPU中d_outputImageRGBA的数据复制到CPU中h_outputImage
    hipMemcpy(h_outputImage,d_outputImageRGBA,sizeof(uchar4)*numPixels,hipMemcpyDeviceToHost);
    Mat output(rows,cols,CV_8UC4,h_outputImage);

    hipFree(d_red);
    hipFree(d_green);
    hipFree(d_blue);
    hipFree(d_redBlurred);
    hipFree(d_greenBlurred);
    hipFree(d_blueBlurred);
    hipFree(d_filter);
    hipFree(d_inputImageRGBA);
    hipFree(d_outputImageRGBA);


    return output;
}

