#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void square(float * d_out,float * d_in)
{
	int idx=threadIdx.x;
	float f=d_in[idx];
	d_out[idx]=f*f;
}
extern "C" int func()
{
	const int array_size=64;
	const int array_bytes=array_size*sizeof(float);
	float h_in[array_size],h_out[array_size];
	for (int i=0;i<array_size;i++)
	{
		h_in[i]=float(i);	
	}
	
	float * d_in;
	float * d_out;
	hipMalloc((void **)&d_in,array_bytes);
	hipMalloc((void **)&d_out,array_bytes);
	
	hipMemcpy(d_in,h_in,array_bytes,hipMemcpyHostToDevice);
	square <<<1,array_size >>> (d_out,d_in);
	hipMemcpy(h_out,d_out,array_bytes,hipMemcpyDeviceToHost);
	
	for(int i=0;i<array_size;i++)
	{
		cout << h_out[i] << endl;
	}
	
	return 0;
}
