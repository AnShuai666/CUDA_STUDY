#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "rgb2gray.cuh"
using namespace std;
using namespace cv;

__global__ void gray(uchar4* d_in,uchar * d_out,int rows, int cols)
{
	int blockId = gridDim.x*blockIdx.y+blockIdx.x;
	int threadId = blockId*blockDim.x*blockDim.y+blockDim.x*threadIdx.y+threadIdx.x;

	if(threadId<rows*cols)
	{
		uchar4 rgba = d_in[threadId];
		d_out[threadId]=0.299f*rgba.x+0.587f*rgba.y+0.114f*rgba.z;

	}
}

//extern "C" 
void func()
{
	Mat src,dst,RGBA;
	src =imread("../../res/1.jpg");
	//imshow("1",src);
    //waitKey(0);
	const int rows=src.rows;
	const int cols=src.cols;
	int num_pixels=rows*cols;
	uchar4 *h_in;
	uchar *h_out;
	uchar4 *d_in;
	uchar *d_out;

	cvtColor(src,RGBA,	CV_BGR2RGBA);
	dst.create(rows,cols,CV_8UC1);
	h_in = (uchar4*)RGBA.ptr<unsigned char>(0);
    h_out = (uchar*)dst.ptr<unsigned char>(0);
	hipMalloc((void**)&d_in, sizeof(uchar4)*num_pixels);
	hipMalloc((void**)&d_out,sizeof(uchar)*num_pixels);
	
	hipMemcpy(d_in,h_in, sizeof(uchar4)*num_pixels,hipMemcpyHostToDevice);
	const dim3 blocksize(32,32,1);
	//const dim3 gridsize((cols-1+blockDim.x)/blockDim.x,((rows-1+blockDim.y)/blockDim.y),1);
    const dim3 gridsize((cols-1+32)/32,((rows-1+32)/32),1);

	gray <<<gridsize,blocksize>>>(d_in,d_out,rows, cols);
	hipMemcpy(h_out,d_out,sizeof(uchar)*num_pixels,hipMemcpyDeviceToHost);




    //unsigned char* data_ptr;
    Mat output(rows,cols,CV_8UC1,h_out);
    
    imwrite("../../output/grayImg.jpg",output);


    hipFree(d_in);
    hipFree(d_out);
    imshow("grayImg",output);
	waitKey(5000);
}
