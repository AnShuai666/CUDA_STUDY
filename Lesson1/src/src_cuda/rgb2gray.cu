#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "rgb2gray.cuh"
using namespace std;
using namespace cv;

__global__ void gray(uchar4* d_in,uchar * d_out,int rows, int cols)
{
	int blockId = gridDim.x*blockIdx.y+blockIdx.x;
	int threadId = blockId*blockDim.x*blockDim.y+blockDim.x*threadIdx.y+threadIdx.x;

	if(threadId<rows*cols)
	{
		uchar4 rgba = d_in[threadId];
		d_out[threadId]=0.299f*rgba.x+0.587f*rgba.y+0.114f*rgba.z;

	}
}

//extern "C"
cv::Mat cudafunc(cv::Mat RGBAImage, cv::Mat GRAYImage)
{

	const int rows=RGBAImage.rows;
	const int cols=RGBAImage.cols;
	int num_pixels=rows*cols;
	uchar4 *h_in;
	uchar *h_out;
	uchar4 *d_in;
	uchar *d_out;

	h_in = (uchar4*)RGBAImage.ptr<unsigned char>(0);
    h_out = (uchar*)GRAYImage.ptr<unsigned char>(0);

	hipMalloc((void**)&d_in, sizeof(uchar4)*num_pixels);
	hipMalloc((void**)&d_out,sizeof(uchar)*num_pixels);
	
	hipMemcpy(d_in,h_in, sizeof(uchar4)*num_pixels,hipMemcpyHostToDevice);
	const dim3 blocksize(32,32,1);
	//const dim3 gridsize((cols-1+blockDim.x)/blockDim.x,((rows-1+blockDim.y)/blockDim.y),1);
    const dim3 gridsize((cols-1+32)/32,((rows-1+32)/32),1);

	gray <<<gridsize,blocksize>>>(d_in,d_out,rows, cols);
	hipMemcpy(h_out,d_out,sizeof(uchar)*num_pixels,hipMemcpyDeviceToHost);

    Mat output(rows,cols,CV_8UC1,h_out);
    hipFree(d_in);
    hipFree(d_out);
    return output;

}
 
