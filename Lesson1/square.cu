
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void square(float * d_out,float * d_in)
{
	int x=threadIdx.x;
	float f=d_in[x];
	d_out[x] =f*f;
}

int main()
{
	const int array_size=64;
	const int array_bytes=array_size*sizeof(float);
	float h_out[array_size],h_in[array_size];
	for(int i=0; i<array_size;i++)
	{
		h_in[i]=float(i);
	}
	
	float * d_out;
	float * d_in;
	hipMalloc((void **)&d_in ,array_bytes);
	hipMalloc((void **)&d_out ,array_bytes);
	
	hipMemcpy(d_in,h_in,array_bytes,hipMemcpyHostToDevice);
	square<<<1,array_size>>>(d_out,d_in);
	hipMemcpy(h_out,d_out,array_bytes,hipMemcpyDeviceToHost);
	
	for(int i=0;i<array_size;i++)
	{
		cout << h_out[i]<< endl;
	}
	return 0;
}
