#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello(void)
{
    printf("Hello World!\n");
}

extern "C" int func()
{
    hello <<<1,10>>>();
    hipDeviceReset();
}

